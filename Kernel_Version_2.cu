#include "hip/hip_runtime.h"
#include <stdio.h>

#include <stdlib.h>

#include "ppm_lib.h"


static void HandleError(hipError_t err,
  const char * file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

# define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))# define CREATOR "PARALLELISME2OPENMP"

PPMImage * readPPM(const char * filename) {
  char buff[16];
  PPMImage * img;
  FILE * fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }
  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }
  if (buff[0] != 'P' || buff[1] != '6') {
    fprintf(stderr, "Invalid image format (must be 'P6')\n");
    exit(1);
  }
  img = (PPMImage * ) malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }
  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n');
    c = getc(fp);
  }
  ungetc(c, fp);
  if (fscanf(fp, "%d %d", & img - > x, & img - > y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }
  if (fscanf(fp, "%d", & rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }
  if (rgb_comp_color != RGB_COMPONENT_COLOR) {
    fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
    exit(1);
  }
  while (fgetc(fp) != '\n');
  img - > data = (PPMPixel * ) malloc(img - > x * img - > y * sizeof(PPMPixel));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }
  if (fread(img - > data, sizeof(PPMPixel) * img - > x, img - > y, fp) != img - > y) {
    fprintf(stderr, "Error loading image '%s'\n", filename);
    exit(1);
  }
  fclose(fp);
  return img;
}
void writePPM(const char * filename, PPMImage * img) {
  FILE * fp;
  fp = fopen(filename, "wb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }
  fprintf(fp, "P6\n");
  fprintf(fp, "# Created by %s\n", CREATOR);
  fprintf(fp, "%d %d\n", img - > x, img - > y);
  fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);
  fwrite(img - > data, 3 * img - > x, img - > y, fp);
  fclose(fp);
}

// Cuda version 2 (avec mémoire partagée)//
// Dans cette version  l'accés à l'image est deplacé de la mémoire globale (cad *img) à la mémoire partagée (en copiant les pixels voulu dans un tableau)
// Pour ca on a copié les pixels que chaque thread d'un block besoin à un tableau en mémoire partagé pour l'utiliser aprés.

__global__ void filterSofter(PPMPixel * img, int * filter, int & divisionFactor, PPMPixel * destination) {

  //un tableau de 3000 Pixel, dans notre cas 6 lignes de l'image.
  __shared__ PPMPixel final[3000];

  //Initialiser les Sommes de RVB pour les diviser sur le facteur du filtre.
  int finalRed = 0;
  int finalGreen = 0;
  int finalBlue = 0;
  // indice pour parcourir le filtre
  int indFiltre = 0;

  //indice du thread dans le block
  int tid = threadIdx.x;
  //indice du thread dans la grille, il nous servira a parcourir toute l'image puisque nbThreads=nbPixels.
  int tidX = threadIdx.x + blockIdx.x * blockDim.x;

  //Pour savoir dans quelle ligne de l'image se situe ce thread
  int ll = tidX / 500;
  //Pour savoir dans quelle colonne de l'image se situe ce thread
  int cc = tidX % 500;

  //c pour savoir dans quelle colonne du block se situe le thread 
  int c = tid % 500;
  int l;

  //chaque thread, dépendamment de sa position, soit il remplit le tableau final avec les deux pixels au dessus soit les deux au dessous
  //à la fin de ce calcul on aura les 6 lignes de l'image remplit dans le tableau final pour l'utiliser au lieu de l'image originale.
  if (tid < 500) {
    final[500 * 2 + c] = img[tidX];
    //Appliquer l'effet mirroir si on accéde a des indices supérieur ou inférieur à la taille de l'image
    if (ll - 1 >= 0)
      final[500 * 1 + c] = img[(ll - 1) * 500 + cc];
    else
      final[500 * 1 + c] = img[(ll + 1) * 500 + cc];

    if (ll - 2 >= 0)
      final[c] = img[(ll - 2) * 500 + cc];
    else
      final[c] = img[(ll + 2) * 500 + cc];

  } else {
    final[500 * 3 + c] = img[tidX];
    //Appliquer l'effet mirroir si on accéde a des indices supérieur ou inférieur à la taille de l'image
    if (ll + 1 > 1000)
      final[500 * 4 + c] = img[(ll + 1) * 500 + cc];
    else
      final[500 * 4 + c] = img[(ll - 1) * 500 + cc];

    if (ll + 2 > 1000)
      final[500 * 5 + c] = img[(ll + 2) * 500 + cc];
    else
      final[500 * 5 + c] = img[(ll - 2) * 500 + cc];

  }
  //pour s'assurer que tous les threads ont fini leurs travaille.
  __syncthreads();

  //on ajouter 1000 au tid pour repositionner l'indice du thread au bon endroit dans le tableau finale
  tid += 1000;
  l = tid / 500;
  c = tid % 500;

  //boucle pour parcourir tout les pixels autour
  for (int i = -2; i <= 2; i++) {
    for (int j = -2; j <= 2; j++) {
      ll = l + i;
      cc = c + j;
      //Appliquer l'effet mirroir si on accéde a des indices supérieur ou inférieur à la taille de l'image
      //cette fois on a pas besoin de tester avec les lignes parce qu'on a deja repositionner l'indice au milieu avec tid+=1000
      if (cc < 0) {
        cc = -cc;
      } else if (cc > 500) {
        cc = c - j;
      }

      //faire la somme des Pixel*Filtre
      finalRed += final[(ll) * 500 + (cc)].red * filter[indFiltre];
      finalGreen += final[(ll) * 500 + (cc)].green * filter[indFiltre];
      finalBlue += final[(ll) * 500 + (cc)].blue * filter[indFiltre];
      indFiltre++;

    }
  }

  //Affecter au pixel l'application du filtre
  destination[tidX].red = finalRed / divisionFactor;
  destination[tidX].green = finalGreen / divisionFactor;
  destination[tidX].blue = finalBlue / divisionFactor;

}

int main() {

  PPMImage * image, * imageCopy;
  image = readPPM("gare_parallelisme.ppm");
  imageCopy = readPPM("gare_parallelisme.ppm");

  /*int HorizontalSobel[25] = { 1,   2,   0,   -2,   -1,
                              4 ,  8,   0 ,  -8 ,  -4,
                              6  , 12 , 0 ,  -12  , -6 ,
                              4,   8,   0 ,  -8,    -4,
                              1,   2,   0,   -2,   -1 };

  int HorizontalSobelDivide=1;*/

  /*int VerticalSobel[25] = { -1,   -4,  -6,   -4,   -1,
                            -2 ,  -8,   -12 ,  -8 ,  -2,
                             0  , 0 , 0 ,  0  , 0 ,
                             2,   8,   12 ,  8,    2,
                             1,   4,   6,   4,   1 };

  int VerticalSobelDivide=1; */
  /*
  int DiagonalShatter[25] = { 1,   0,   0,   0,  1,
                             0 ,  0,   0 ,  0 ,  0,
                             0  , 0 , 0 ,  0  , 0 ,
                             0,   0,   0 ,  0,    0,
                             1,   0,   0,   0,   1 };

  int DiagonalShatterDivide=4;*/

  int HorizontalBlur[25] = {
    0,
    0,
    0,
    0,
    0,
    0,
    0,
    0,
    0,
    0,
    1,
    2,
    3,
    2,
    1,
    0,
    0,
    0,
    0,
    0,
    0,
    0,
    0,
    0,
    0
  };

  int HorizontalBlurDivide = 9;
  /*
  int Soften[25] = { 1,   1,   1,   1,   1,
                     1 ,  1,   1 ,  1 ,  1,
                     1  , 1 , 1 ,  1  , 1 ,
                     1,   1,   1 ,  1,    1,
                     1,   1,   1,   1,   1 };
  int SoftenDivide=25;*/

  /*int SharpenMeduim[25] = { -1,   -1,   -1,   -1,  -1,
                            -1 ,  -1,   -1 ,  -1 , -1,
                            -1, -1, 49 , -1  , -1,
                            -1,   -1,   -1,  -1, -1,
                            -1,   -1,   -1,   -1, -1 };
  int SharpenMeduimDivide=25;
  */

  PPMPixel * dev_image;
  PPMPixel * dev_imageCopy;
  int * dev_filter;
  int * dev_divisionFactor;

  float time;
  hipEvent_t start, stop;

  HANDLE_ERROR(hipMalloc((void ** ) & dev_image, image - > x * image - > y * 3 * sizeof(char)));
  HANDLE_ERROR(hipMalloc((void ** ) & dev_imageCopy, imageCopy - > x * imageCopy - > y * 3 * sizeof(char)));
  HANDLE_ERROR(hipMalloc((void ** ) & dev_filter, 25 * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void ** ) & dev_divisionFactor, sizeof(int)));

  HANDLE_ERROR(hipMemcpy(dev_image, image - > data, image - > x * image - > y * 3 * sizeof(char), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_imageCopy, imageCopy - > data, imageCopy - > x * imageCopy - > y * 3 * sizeof(char), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_filter, HorizontalBlur, 25 * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_divisionFactor, & HorizontalBlurDivide, sizeof(int), hipMemcpyHostToDevice));

  hipEventCreate( & start);
  hipEventCreate( & stop);
  hipEventRecord(start, 0);

  printf(">%s\n", hipGetErrorString(hipGetLastError()));

  for (int i = 0; i < 1000; i++) {
    filterSofter << < 500, 1000 >>> (dev_image, dev_filter, * dev_divisionFactor, dev_imageCopy);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( & time, start, stop);

  HANDLE_ERROR(hipMemcpy(imageCopy - > data, dev_imageCopy, imageCopy - > x * imageCopy - > y * 3 * sizeof(char), hipMemcpyDeviceToHost));

  printf("Temps nécessaire :  %3.1f ms\n", time);

  writePPM("Result_Version_2.ppm", imageCopy);

  /* liberer la memoire allouee sur le GPU */
  hipFree(dev_image);
  hipFree(dev_imageCopy);
  hipFree(dev_filter);

  return 0;
}